#include "hip/hip_runtime.h"
__global__ void reduceCompleteUnrollWarps8(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * k;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int a5 = g_idata[idx + 4 * blockDim.x];
        int a6 = g_idata[idx + 5 * blockDim.x];
        int a7 = g_idata[idx + 6 * blockDim.x];
        int a8 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512)
        idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256)
        idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128)
        idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64)
        idata[tid] += idata[tid + 64];
    __syncthreads();

    if (tid < 32)
    {
        int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        __syncthreads();
        vmem[tid] += vmem[tid + 16];
        __syncthreads();
        vmem[tid] += vmem[tid + 8];
        __syncthreads();
        vmem[tid] += vmem[tid + 4];
        __syncthreads();
        vmem[tid] += vmem[tid + 2];
        __syncthreads();
        vmem[tid] + vmem[tid + 1];
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = idata[0];
}
#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define run_kernel(s)                                                                     \
    iStart = seconds();                                                                   \
    mathKernel##s<<<grid, block>>>(d_C);                                                  \
    hipDeviceSynchronize();                                                              \
    iElaps = seconds() - iStart;                                                          \
    printf("matchKernel%s <<< %4d %4d >>> elapsed %f sec \n", #s,grid.x, block.x, iElaps); \
    CHECK(hipGetLastError()); 

__global__ void mathKernel1(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    float a, b; 
    a = b = 0.0f; 
    if (tid % 2 == 0) {
        a = 100.0f; 
    } 
    else {
        b = 200.0f;
    }
    c[tid] = a + b; 
}

__global__ void mathKernel2(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    float a,b; 
    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0) {
        a = 100.0f; 
    }
    else {
        b = 200.0f; 
    }
    c[tid] = a + b; 
}

__global__ void mathKernel3(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    float ia, ib; 
    ia = ib = 0.0f; 

    bool ipred = (tid % 2 == 0);
    if (ipred) {
        ia = 100.0f; 
    }
    else {
        ib = 200.0f; 
    }
    c[tid] = ia + ib; 
}

__global__ void warmingup(float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    float a,b; 
    a = b = 0.0f;

    if ((tid / warpSize) % 2 == 0) {
        a = 100.0f; 
    }
    else {
        b = 200.0f; 
    }
    c[tid] = a + b;     
}



int main(int argc, char** argv)
{
    // set up device
    int dev = 0; 
    hipDeviceProp_t deviceProp; 
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

    // set up data size
    int size = 64; 
    int blocksize = 6; 
    if (argc > 1) blocksize = atoi(argv[1]);
    if (argc > 2) size = atoi(argv[2]);
    printf("Data size %d ", size);

    // set up execution configuration
    dim3 block(blocksize, 1);
    dim3 grid((size+block.x-1)/block.x,1);
    printf("Execution Configure (block %d grid %d)\n",block.x, grid.x);

    // allocate gpu memory
    float *d_C;
    size_t nBytes = size * sizeof(float);
    hipMalloc((float**)&d_C, nBytes);

    // run a warmup kernel to remove overhead
    size_t iStart, iElaps; 
    hipDeviceSynchronize();
    iStart = seconds();
    warmingup<<<grid, block>>>(d_C);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("warmup      <<< %4d %4d >>> elapsed %d sec \n", grid.x,block.x, iElaps);

    // run kernel 1
    run_kernel(1);

    // run kernel 2
    run_kernel(2);

    // run kernel 3
    run_kernel(3);

    // run kernel 4
    // run_kernel(4);

    // free gpu memory and reset divece
    hipFree(d_C); 
    hipDeviceReset();
    return EXIT_SUCCESS;
}
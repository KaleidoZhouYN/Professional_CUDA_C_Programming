
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU!\n");
}

int main(void)
{
    // hello from cpu
    printf("Hello World form CPU!\n");

    helloFromGPU<<<1, 10>>>(); 
    hipDeviceReset(); 
    return 0; 
}

/*
the output is:
Hello World form CPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
Hello World from GPU!
*/
int main(int argc, char **argv) {
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    hipSetDevice(dev);
    // set up array size
    int nElem = 1<<20; // total number of elements to reduce
    printf(" with array size %d\n", nElem);
    size_t nBytes = nElem * sizeof(float);
    // set up offset for summary
    int blocksize = 512;
    int offset = 0;
    if (argc>1) offset = atoi(argv[1]);
    if (argc>2) blocksize = atoi(argv[2]);
    // execution configuration
    dim3 block (blocksize,1);
    dim3 grid ((nElem+block.x-1)/block.x,1);
    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *hostRef = (float *)malloc(nBytes);
    float *gpuRef = (float *)malloc(nBytes);
    // initialize host array
    initialData(h_A, nElem);
    memcpy(h_B,h_A,nBytes);
    // summary at host side
    sumArraysOnHost(h_A, h_B, hostRef,nElem,offset);
    // allocate device memory
    float *d_A,*d_B,*d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);
    // copy data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_A, nBytes, hipMemcpyHostToDevice);
    // kernel 1:
    double iStart = seconds();
    warmup <<< grid, block >>> (d_A, d_B, d_C, nElem, offset);
    hipDeviceSynchronize();
    double iElaps = seconds() - iStart;
    printf("warmup <<< %4d, %4d >>> offset %4d elapsed %f sec\n",
    grid.x, block.x,
    offset, iElaps);
    iStart = seconds();
    readOffset <<< grid, block >>> (d_A, d_B, d_C, nElem, offset);
    hipDeviceSynchronize();
    iElaps = seconds() - iStart;
    printf("readOffset <<< %4d, %4d >>> offset %4d elapsed %f sec\n",
    grid.x, block.x,
    offset, iElaps);
    // copy kernel result back to host side and check device results
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    checkResult(hostRef, gpuRef, nElem-offset);
    // copy kernel result back to host side and check device results
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    checkResult(hostRef, gpuRef, nElem-offset);
    // copy kernel result back to host side and check device results
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    checkResult(hostRef, gpuRef, nElem-offset);
    // free host and device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);

     // reset device
    hipDeviceReset();
    return EXIT_SUCCESS;
}
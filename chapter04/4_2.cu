#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ float devData; 

__global__ void checkGlobalVariable() {
    // display the original value
    printf("Device: the value of the global variable is %f\n", devData);

    // alter the values
    devData += 2.0f; 
}

int main(void) {
    // initialize the global variable
    float value = 3.14f; 

    float* devPtr; 
    hipGetSymbolAddress((void**)&devPtr,HIP_SYMBOL(devData));
    hipMemcpy(devPtr, &value, sizeof(float) , hipMemcpyHostToDevice);
    printf("Host:   cpoied %f to the global variable\n", value);

    // invoke the kernel
    checkGlobalVariable<<<1,1>>>();

    // copy the global variable back to the host
    hipMemcpy(&value, devPtr, sizeof(float), hipMemcpyDeviceToHost);
    printf("Host:   the value changed by the kernel to %f\n", value);

    hipDeviceReset();
    return EXIT_SUCCESS;
}
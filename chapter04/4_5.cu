#include <hip/hip_runtime.h>
#include<stdio.h>

void sumArraysOnHost(float *a, float*b, float*c, int nElem, int offset) {
    for (int i = 0; i < nElem; i++) {
        unsigned int k = i + offset; 
        if (k < nElem) c[i] = a[k] + b[k];
    }
}

__global__ void sumArrays(float* a, float* b, float* c, int nElem, int offset) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int k = idx + offset; 
    if (k >= nElem)
        return; 

    c[idx] = a[k] + b[k];
}

__global__ void sumArraysZeroCopy(float* a, float* b, float* c, int nElem, int offset) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int k = idx + offset; 
    if (k >= nElem)
        return; 

    c[idx] = a[k] + b[k];
}

void initialData(float* a, int nElem) {
    memset(a, 1, sizeof(nElem));
}

void checkResult(float* hostRef, float* gpuRef, int nElem) {
    bool equal = 1;
    for (int i = 0; i < nElem; i++)
        if (hostRef[i] != gpuRef[i]) {
            equal = 0; 
            break; 
        }
    printf("Host equal to gpu, %d",equal);
}

int main(int argc, char** argv) {
    // part 0 : set up device and array
    // set up device
    int dev = 0; 
    hipSetDevice(dev);

    // get device properties
    hipDeviceProp_t deviceProp; 
    hipGetDeviceProperties(&deviceProp, dev);

    // check if support mapped memory
    if (!deviceProp.canMapHostMemory) {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    printf("Using Device %d: %s ", dev, deviceProp.name);

    // set up date size of vectors
    int ipower = 10; 
    if (argc>1) ipower = atoi(argv[1]);
    int nElem = 1<<ipower; 
    size_t nBytes = nElem * sizeof(float);
    if (ipower < 18) {
        printf("Vector size %d power %d nbytes %3.0f KB\n", nElem, \
            ipower, (float)nBytes/(1024.f));
    }
    else {
        printf("Vector size %d power %d nbytes %3.0f MB\n", nElem, \
            ipower, (float)nBytes/(1024.0f*1024.f));
    }

    int offset = 0; 
    if (argc > 2) offset = atoi(argv[2]);

    // part 1: using device memory
    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef; 
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);

    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);

    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // set up execution configuration
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid ((nElem+block.x-1)/block.x);

    // invoke kernel at host side
    sumArrays <<<grid, block>>>(d_A, d_B, d_C, nElem, offset);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);

    // part 2: using zerocopy memory for array A and B
    // allocate zerocpy memory
    unsigned int flags = hipHostMallocMapped;
    hipHostAlloc((void **)&h_A, nBytes, flags);
    hipHostAlloc((void **)&h_B, nBytes, flags);

    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // pass the pointer to device
    hipHostGetDevicePointer((void **)&d_A, (void *)h_A, 0);
    hipHostGetDevicePointer((void **)&d_B, (void *)h_B, 0);

    // add at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem, offset);    

    // execute kernel with zero copy memory
    sumArraysZeroCopy <<<grid, block>>>(d_A, d_B, d_C, nElem, offset);

    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free memory
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    hipDeviceReset();
    return EXIT_SUCCESS;
}